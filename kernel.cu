
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <omp.h>
#include <math.h>       /* fabsf */
#include <string.h>
#include <stdlib.h>
#include <time.h>

#define DEBUG 1
#define NO_BLOCKS 256
#define NO_THREADS 32
//Error check-----
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
//Error check-----
//This is a very good idea to wrap your calls with that function.. Otherwise you will not be able to see what is the error.
//Moreover, you may also want to look at how to use cuda-memcheck and cuda-gdb for debugging.

__global__ void rowSum(int *adj, int *xadj, int* tadj, int* txadj,double *rv, double *cv,int *nov,double *errorOut)
{
	int step = 32*1024;
	int  index = blockDim.x * blockIdx.x + threadIdx.x;
        for(int i = index; i<(*nov)-1; i+=step){
                double  rsum = 0;
                int  row_start  = xadj[i];
                int  row_end    = xadj[i +1];
                for (int jj = row_start; jj < row_end; jj++)
                        rsum += cv[adj[jj]];
                if(rsum != 0)
                {
                        rv[i] = 1.0 / rsum;
                }

        }

}

__global__ void colSum(int *tadj, int *txadj, double *rv, double *cv,int *nov)
{
	int step = 32*1024;
	int  index = blockDim.x * blockIdx.x + threadIdx.x;
	for(int i = index; i<(*nov)-1; i+=step){
                double  csum = 0;
                int  col_start  = txadj[i];
                int  col_end    = txadj[i +1];
                for (int jj = col_start; jj < col_end; jj++)
                        csum += rv[tadj[jj]];
                if(csum != 0)
                {
                        cv[i] = 1.0 / csum;
                }

        }
        __syncthreads();

}

__global__ void errorCheck(int *adj, int *xadj, double *rv, double *cv, int *nov, double *errorOut)
{
	int step = 32*1024;
	__shared__ double sharedMem[1024];
	int  index = blockDim.x * blockIdx.x + threadIdx.x;
	for(int i = index; i<(*nov)-1; i+=step)
        {
                double errorSum = 0;
                int row_start = xadj[i];
                for(int jj = row_start; jj<xadj[i+1]; jj++)
                        errorSum += (cv[adj[jj]] * rv[i]);
                double errorVal = fabsf(1.0 - errorSum);
                sharedMem[threadIdx.x] = sharedMem[threadIdx.x] < errorVal ? errorVal :sharedMem[threadIdx.x];
        }
		__syncthreads();
	int ib = blockDim.x / 2;
	while(ib != 0)
	{
		if(threadIdx.x < ib && sharedMem[threadIdx.x + ib] > sharedMem[threadIdx.x])
		{
			sharedMem[threadIdx.x] = sharedMem[threadIdx.x + ib];
		}
		__syncthreads();
		ib /=2;
	}
	if(threadIdx.x == 0)
		errorOut[blockIdx.x] = sharedMem[0];
	sharedMem[threadIdx.x] = 0;	

}


void wrapper(int* adj, int* xadj, int* tadj, int*txadj, double* rv, double* cv, int* nov, int* nnz, int siter){
  
  printf("Wrapper here! \n");
  
  int *d_adj, *d_xadj, *d_tadj, *d_txadj, *d_nov, *d_nnz, *d_siter;
  double *d_rv,*d_cv, *d_errorOut, *d_errorSortedOut, *errorOut;

  double errorValue;  
  
  errorOut = (double*)malloc(512 * sizeof(double));

  hipMalloc(&d_adj, *nnz * sizeof(int));
  hipMalloc(&d_tadj, *nnz * sizeof(int));
  hipMalloc(&d_xadj, *nov * sizeof(int)); 
  hipMalloc(&d_txadj, *nov * sizeof(int));

  hipMalloc(&d_nov, sizeof(int));
  hipMalloc(&d_nnz, sizeof(int));
  hipMalloc(&d_siter, sizeof(int));

  hipMalloc(&d_rv, *nov * sizeof(double));
  hipMalloc(&d_cv, *nov * sizeof(double));	

  hipMalloc(&d_errorOut, *nov * sizeof(double));
  hipMalloc(&d_errorSortedOut, *nov * sizeof(double));

  hipMemcpy(d_adj, adj, *nnz*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_tadj, tadj, *nnz*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_xadj, xadj, *nov*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_txadj, txadj, *nov*sizeof(int), hipMemcpyHostToDevice);
   
  hipMemcpy(d_nov, nov, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_nnz, nnz, sizeof(int), hipMemcpyHostToDevice);
  //cudaMemcpy(d_siter, &siter, sizeof(int), cudaMemcpyHostToDevice);

  hipMemcpy(d_rv, rv, *nov*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_cv, cv, *nov*sizeof(double), hipMemcpyHostToDevice);
 
  int size = (*nov * sizeof(double));
  errorOut = (double*)malloc(size);
  hipEvent_t start,stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventRecord(start, 0);
 
  for(int i = 0; i<siter; i++)
  {
	rowSum<<<32,1024>>>(d_adj, d_xadj,d_tadj,d_txadj, d_rv, d_cv, d_nov,d_errorOut);
  	gpuErrchk( hipDeviceSynchronize() );

  	colSum<<<32,1024>>>(d_tadj,d_txadj, d_rv, d_cv, d_nov);
  	gpuErrchk( hipDeviceSynchronize() );

  	errorCheck<<<32,1024>>>(d_adj, d_xadj, d_rv, d_cv, d_nov, d_errorOut);
  	gpuErrchk( hipDeviceSynchronize() );
	
  	hipMemcpy(errorOut, d_errorOut, 32*sizeof(double), hipMemcpyDeviceToHost);
	errorValue = 0;
	for(int k = 0; k<32; k++)
	{
		if(errorValue < errorOut[k])
		{
			errorValue = errorOut[k];
		}	
	}

  	printf("iter-%i error : %f \n\n",i ,errorValue);
	hipMemset(d_errorOut, 0, *nov*sizeof(double));
  } 
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("GPU scale took: %f s\n", elapsedTime/1000);
  free(errorOut);
  hipFree(d_adj);
  hipFree(d_xadj);
  hipFree(d_tadj);
  hipFree(d_txadj);
  hipFree(d_errorOut);
  hipFree(d_errorSortedOut);
  hipFree(d_rv);
  hipFree(d_cv);
  hipFree(d_nov);
  hipFree(d_nnz);
    
}


